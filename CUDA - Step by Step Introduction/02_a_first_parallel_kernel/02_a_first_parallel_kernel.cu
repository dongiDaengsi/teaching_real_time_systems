
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int *a, int *b, int *c) {
  //c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
  c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}

#define N 512
int main(void) {
  int *a, *b, *c;       // host copies of a, b, c
  int *d_a, *d_b, *d_c; // device copies of a, b, c
  int size = N * sizeof(int);

  // Alloc space for device copies of a, b, c
  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);

  // Alloc space for host copies of a, b, c and setup input values
  a = (int *)malloc(size);
  b = (int *)malloc(size);
  c = (int *)malloc(size);
  for (int i=0; i<N; i++)
  {
    a[i] = 1;
    b[i] = 4;
    c[i] = 0;
  }

  // Copy inputs to device
  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

  // Launch add() kernel on GPU with N blocks
  //add<<<N,1>>>(d_a, d_b, d_c);

  // Launch add() kernel on GPU with N threads
  add<<<1,N>>>(d_a, d_b, d_c);

  // Copy result back to host
  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

  printf("Resulting vector is:\n");
  for (int i=0; i<N; i++)
  {
    printf("%d ", c[i]);
  }
  printf("\n");

  // Cleanup
  free(a); free(b); free(c);
  hipFree(d_a); hipFree(d_b); hipFree(d_c);
  return 0;
}
