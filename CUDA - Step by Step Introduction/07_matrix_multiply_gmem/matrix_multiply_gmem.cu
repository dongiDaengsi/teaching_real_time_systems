// file: matrix_multiply_gmem.cu
//
// shows how to multiply two matrices A,B
// by making use of a GPU
//



#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
using namespace std;

namespace help {
    template <class T>
    inline void print_array1d(T a, int n){
        for(int i=0;i<n;++i){
            std::cout<<a[i]<<' ';
        }
        std::cout<<'\n';
    }

    template <class T>
    inline void print_array2d(string str, T a, int n, int m){
        std::cout << str << ":\n";
        for(int i=0;i<n;++i){
            for(int j=0; j<m; ++j){
                std::cout<<a[j+i*m]<<' ';
            }
            std::cout<<'\n';
        }
    }

} // namespace help


// Matrices are stored in row-major order:
// M(row, col) = *(M.elements + row * M.width + col)

typedef struct {
    int width;
    int height;
    float* elements;
} Matrix;

// Thread block size
#define BLOCK_SIZE 32


// Matrix multiplication kernel called by MatMul()
__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C)
{
    // Each thread computes one element of C
    // by accumulating results into Cvalue
    float Cvalue = 0;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if(row>=C.height||col>=C.width)
        return;
    for (int e = 0; e < A.width; ++e)
        Cvalue += A.elements[row * A.width + e]
                * B.elements[e * B.width + col];
    C.elements[row * C.width + col] = Cvalue;
}


// Matrix multiplication - Host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE
void MatMul(const Matrix A, const Matrix B, Matrix C)
{
    // 1. Copy matrix A to device memory
    Matrix d_A;
    d_A.width = A.width; d_A.height = A.height;
    size_t size = A.width * A.height * sizeof(float);
    hipMalloc(&d_A.elements, size);
    hipMemcpy(d_A.elements, A.elements, size,
               hipMemcpyHostToDevice);

    // 2. Copy matrix B to device memory
    Matrix d_B;
    d_B.width = B.width; d_B.height = B.height;
    size = B.width * B.height * sizeof(float);
    hipMalloc(&d_B.elements, size);
    hipMemcpy(d_B.elements, B.elements, size,
               hipMemcpyHostToDevice);

    // 3. Allocate device memory for matrix C
    Matrix d_C;
    d_C.width = C.width; d_C.height = C.height;
    size = C.width * C.height * sizeof(float);
    hipMalloc(&d_C.elements, size);

    // 4. Setup configuration

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

    int gdimx = (B.width+dimBlock.x-1) / dimBlock.x;
    int gdimy = (A.height+dimBlock.y-1) / dimBlock.y;
    dim3 dimGrid(gdimx, gdimy);

    printf("dimBlock(%d,%d)\n", BLOCK_SIZE,BLOCK_SIZE);
    printf("dimGrid(%d,%d)\n", gdimx, gdimy );

    // 5. Launch the kernel
    MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);

    // 6. Copy result matrix from device memory to host memory
    hipMemcpy(C.elements, d_C.elements, size,
               hipMemcpyDeviceToHost);

    // 7. Free device memory
    hipFree(d_A.elements);
    hipFree(d_B.elements);
    hipFree(d_C.elements);
}



int main(){

    // 1. prepare memory for host matrix A
    Matrix A, B, C;
    A.height=100; A.width=100;
    size_t N_A=A.width * A.height;
    size_t sizeA = A.width * A.height * sizeof(float);
    A.elements=(float *) malloc(sizeA);

    // 2. prepare memory for host matrix B
    B.height=A.width; B.width=100;
    size_t N_B=B.width * B.height;
    size_t sizeB = B.width * B.height * sizeof(float);
    B.elements=(float *) malloc(sizeB);

    // 3. prepare memory for host matrix C
    C.height=A.height; C.width=B.width;
    size_t N_C=C.width * C.height;
    // size_t sizeC = C.width * C.height * sizeof(float);
    // C.elements=(float *) malloc(sizeC);
    C.elements = new float[N_C]();

    // 4. initialize matrix A and B
    for(int i=0;i<N_A;++i){
        A.elements[i]=2.0;
    }
    for(int i=0;i<N_B;++i){
        B.elements[i]=5.0f;
    }

    // 5. print input matrices (if matrices are small)
    if ((A.height<=20) && (A.width<=20) && (B.width<=20))
    {
      help::print_array2d<float *>("A", A.elements,A.height, A.width);
      help::print_array2d<float *>("B", B.elements,B.height, B.width);
      help::print_array2d<float *>("C", C.elements,C.height, C.width);
    }

    // 6. do the matrix multiplication on the GPU
    MatMul(A, B, C);

    // 7. print multiplication result matrix C (if matrix is small))
    if ((C.height<=20) && (C.width<=20))
      help::print_array2d<float *>("C", C.elements,C.height, C.width);

    // 8. free host memory
    free(A.elements);free(B.elements);
    delete[] C.elements;

} // main
