
#include <hip/hip_runtime.h>
#include <stdio.h>

const int N_in=86;
const int RADIUS=3;

const int N_out=N_in-2*RADIUS;

const int NR_BLOCKS  = 8;
const int NR_THREADS = 10;



__global__ void stencil_1d(int *in, int *out) {

 // 1. define shared memory
 __shared__ int temp[NR_THREADS + 2 * RADIUS];

 // 2. compute global (in-array) and
 //    local array (temp-array) indices
 int gindex = threadIdx.x + blockIdx.x * blockDim.x + RADIUS;
 int lindex = threadIdx.x + RADIUS;

 // 3. Read one more input element from
 // input array <in> into shared memory
 temp[lindex] = in[gindex];

 // 4. Make sure, also the "border" elements
 //    are read in
 if (threadIdx.x < RADIUS) {
   temp[lindex - RADIUS]     = in[gindex - RADIUS];
   temp[lindex + NR_THREADS] = in[gindex + NR_THREADS];
 }

// 5. Make sure, data is already copied
//    completely to <temp> buffer, in order
//    to compute stencil in next step
 __syncthreads();

 // 6. Compute stencil result
 int result = 0;
 for (int offset = -RADIUS ; offset <= RADIUS ; offset++)
  result += temp[lindex + offset];

 // 7. Store the result
 //printf("Thread %d in block %d (gindex=%d) has computed result=%d\n",
 //        threadIdx.x, blockIdx.x, gindex, result);
 out[gindex - RADIUS] = result;
}



int main(void) {

  // 1. Prepare host and device memory structures
  int *in, *out;     // host memory
  int *d_in, *d_out; // device memory
  int size_in  = N_in  * sizeof(int);
  int size_out = N_out * sizeof(int);


  // 2. Alloc space for device memory
  hipMalloc((void **)&d_in, size_in);
  hipMalloc((void **)&d_out, size_out);


  // 3. Alloc space for host memory
  in  = (int *)malloc(size_in);
  out = (int *)malloc(size_out);
  for (int i=0; i<N_in; i++)
    in[i] = 1;

  for (int i=0; i<N_out; i++)
    out[i] = 0;



  // 4. Copy host memory to device memory
  hipMemcpy(d_in,   in, size_in,  hipMemcpyHostToDevice);
  hipMemcpy(d_out, out, size_out, hipMemcpyHostToDevice);


  // 5. Launch stencil_1d() kernel on GPU
  printf("NR_BLOCKS:  %d\n", NR_BLOCKS);
  printf("NR_THREADS: %d\n", NR_THREADS);
  stencil_1d<<<NR_BLOCKS,NR_THREADS>>>(d_in, d_out);


  // 6. Copy result back from device to host
  hipMemcpy(out, d_out, size_out, hipMemcpyDeviceToHost);


  // 7. Show result (output) array
  printf("\nHere is the result:\n");
  for (int i=0; i<N_out; i++)
  {
    printf("%d ", out[i]);
  }
  printf("\n");

  // 8. Cleanup host and device memory
  free(in); free(out);
  hipFree(d_in); hipFree(d_out);


  // 9. Finished!
  return 0;
}
