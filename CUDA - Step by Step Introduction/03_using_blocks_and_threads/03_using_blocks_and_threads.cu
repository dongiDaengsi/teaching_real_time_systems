
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N (1024*2048)
#define THREADS_PER_BLOCK 512


__global__ void add(int *a, int *b, int *c, int n) {
 int index = threadIdx.x + blockIdx.x * blockDim.x;
 c[index] = a[index]+b[index];
}

int main(void) {

  int NrBlocks = N/THREADS_PER_BLOCK;
  printf("Launching kernel with:\n");
  printf("\t NrBlocks: %d\n", NrBlocks);
  printf("\t THREADS_PER_BLOCK: %d\n", THREADS_PER_BLOCK);

  int *a, *b, *c;       // host copies of a, b, c
  int *d_a, *d_b, *d_c; // device copies of a, b, c
  int size = N * sizeof(int);

  // Alloc space for device copies of a, b, c
  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);

  // Alloc space for host copies of a, b, c and setup input values
  printf("Preparing vectors of size %d\n", N);
  a = (int *)malloc(size);
  b = (int *)malloc(size);
  c = (int *)malloc(size);
  for (int i=0; i<N; i++)
  {
    a[i] = 1;
    b[i] = 4;
    c[i] = 0;
  }

  // Copy inputs to device
  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);


  // Launch add() kernel on GPU with several
  // blocks and THREADS_PER_BLOCK many threads
  // per block
  add<<<NrBlocks,THREADS_PER_BLOCK>>>(d_a, d_b, d_c, N);


  // Copy result back to host
  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

  printf("\nFirst 10 elements of resulting vector are:\n");
  for (int i=0; i<10; i++)
  {
    printf("%d ", c[i]);
  }
  printf("\nLast 10 elements of resulting vector are:\n");
  for (int i=N-10; i<N; i++)
  {
    printf("%d ", c[i]);
  }
  printf("\n");

  // Cleanup
  free(a); free(b); free(c);
  hipFree(d_a); hipFree(d_b); hipFree(d_c);
  return 0;
}
